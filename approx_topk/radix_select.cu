#include "hip/hip_runtime.h"
// radix-select top-k implementation.
// Largely copied from Pytorch:
// https://github.com/pytorch/pytorch/blob/main/aten/src/ATen/native/cuda/TensorTopK.cu
// ...but with the multi-block code removed.

// #define TORCH_ASSERT_NO_OPERATORS
#include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/ScanUtils.cuh>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/SortingRadixSelect.cuh>

#include <torch/extension.h>

#include <c10/macros/Macros.h>

namespace pytorch_topk
{
  using namespace at;
  using namespace at::native;

  template <typename T>
  struct AddOp
  {
    __device__ __forceinline__ T operator()(T const &lhs, T const &rhs)
    {
      return (lhs + rhs);
    }
  };

  template <typename T, typename IndexType, int Dim>
  C10_LAUNCH_BOUNDS_1(1024)
  __global__ void gatherTopK(at::cuda::detail::TensorInfo<const T, IndexType> input,
                             IndexType inputSliceSize,
                             IndexType outputSliceSize, // aka `k`
                             bool largest,

                             IndexType numInputSlices,
                             IndexType inputWithinSliceStride,

                             at::cuda::detail::TensorInfo<T, IndexType> topK,
                             IndexType topKWithinSliceStride,

                             at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
                             IndexType indicesWithinSliceStride)
  {
    // Indices are limited to integer fp precision, so counts can fit in
    // int32, regardless of IndexType
#if defined(USE_ROCM)
    __shared__ int smem[64];
#else
    __shared__ int smem[32]; // one per each warp, up to warp limit
#endif
    IndexType slice = at::native::getLinearBlockId<IndexType>();
    if (slice >= numInputSlices)
    {
      return;
    }

    // Find the start offset for our slice
    IndexType sliceStartIndex =
        at::cuda::detail::IndexToOffset<const T, IndexType, Dim>::get(slice, input);
    IndexType topKSliceStartIndex =
        at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, topK);
    IndexType indicesSliceStartIndex =
        at::cuda::detail::IndexToOffset<int64_t, IndexType, Dim>::get(slice, indices);

    const T *inputSliceStart = &input.data[sliceStartIndex];
    T *topKSliceStart = &topK.data[topKSliceStartIndex];
    int64_t *indicesSliceStart = &indices.data[indicesSliceStartIndex];

    // Find the k-th highest element in our input
    T topKValue;
    topKValue = static_cast<T>(0);
    radixSelect<T, typename TopKTypeConfig<T>::RadixType, IndexType>(
        inputSliceStart, outputSliceSize, largest,
        inputSliceSize, inputWithinSliceStride,
        smem, &topKValue);
    const auto topKConverted = at::native::TopKTypeConfig<T>::convert(topKValue);

    // Every value that is strictly less/greater than `pattern`
    // (depending on sort dir) in sorted int format is in the top-K.
    // The top-K value itself might not be unique.
    //
    // Since there are a variable number of elements that we see that
    // are within the top-k, we don't know at what index to write out
    // the resulting values.
    // In order to get this, we perform an exclusive prefix sum of
    // `hasTopK`. This will return the resulting index into which we
    // need to write the result, if a thread has a result.

    // All threads need to participate in the loop and the prefix sum,
    // but not necessarily in the load; hence loop bounds being rounded
    // up to a multiple of the block dim.
    IndexType numIterations = round_up(inputSliceSize, (IndexType)blockDim.x);
    IndexType writeIndexStart = 0;

    for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x)
    {
      bool inRange = (i < inputSliceSize);
      T v =
          inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
      const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
      bool hasTopK;
      if (largest)
      {
        hasTopK = inRange && (convertedV > topKConverted);
      }
      else
      {
        hasTopK = inRange && (convertedV < topKConverted);
      }

      int index;
      int carry;
      at::cuda::exclusiveBinaryPrefixScan<int, true>(
          smem, hasTopK, &index, &carry, AddOp<int>());

      if (hasTopK)
      {
        int writeIndex = writeIndexStart + index;
        CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

        IndexType topKOffset = writeIndex * topKWithinSliceStride;
        IndexType indexOffset = writeIndex * indicesWithinSliceStride;

        topKSliceStart[topKOffset] = v;
        indicesSliceStart[indexOffset] = i;
      }

      writeIndexStart += carry;
    }

    // We need to fill in the rest with actual == top-K values.
    // The number that we need is outputSliceSize -
    // writeIndexStart. There might be more than that number available,
    // in which case we have to choose the first seen set. We do this
    // via a prefix sum to calculate indices for writing results.
    CUDA_KERNEL_ASSERT(outputSliceSize >= writeIndexStart);
    IndexType topKRemaining = (outputSliceSize - writeIndexStart);

    for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x)
    {
      bool inRange = (i < inputSliceSize);
      T v =
          inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
      const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
      bool hasTopK = inRange && (convertedV == topKConverted);

      int index;
      int carry;
      at::cuda::exclusiveBinaryPrefixScan<int, true>(
          smem, hasTopK, &index, &carry, AddOp<int>());

      if (hasTopK && index < topKRemaining)
      {
        int writeIndex = writeIndexStart + index;
        CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

        IndexType topKOffset = writeIndex * topKWithinSliceStride;
        IndexType indexOffset = writeIndex * indicesWithinSliceStride;

        topKSliceStart[topKOffset] = v;
        indicesSliceStart[indexOffset] = i;
      }

      if (carry >= topKRemaining)
      {
        break;
      }

      topKRemaining -= carry;
      writeIndexStart += carry;
    }
  };

  template <typename T, typename IndexType, int Dim>
  void launch(
      at::cuda::detail::TensorInfo<const T, IndexType> input,
      IndexType inputSliceSize,
      IndexType outputSliceSize, // aka `k`
      bool largest,

      IndexType numInputSlices,
      IndexType inputWithinSliceStride,

      at::cuda::detail::TensorInfo<T, IndexType> topK,
      IndexType topKWithinSliceStride,

      at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
      IndexType indicesWithinSliceStride)
  {

    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(numInputSlices, grid), "Too many slices for topk");
    int warp_size = at::cuda::warp_size();
    dim3 block(std::min(at::ceil_div((int64_t)inputSliceSize, (int64_t)warp_size) * (int64_t)warp_size, (int64_t)1024));
    gatherTopK<T, IndexType, Dim><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
        input,
        inputSliceSize,
        outputSliceSize,
        largest,
        numInputSlices,
        inputWithinSliceStride,
        topK,
        topKWithinSliceStride,
        indices,
        indicesWithinSliceStride);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  void launch_gather_topk_kernel(
      const Tensor &self, int64_t k, int64_t dim, bool largest,
      const Tensor &values, const Tensor &indices)
  {
    // checkAllSameGPU(__func__, {topK_arg, indices_arg, input_arg});
    // TORCH_CHECK(self.dtype() == at::kFloat);
    // TORCH_CHECK(b.dtype() == at::kFloat);
    // TORCH_INTERNAL_ASSERT(a.device().type() == at::DeviceType::CUDA);
    // TORCH_INTERNAL_ASSERT(b.device().type() == at::DeviceType::CUDA);

    int numDims = self.dim();
    numDims = numDims == 0 ? 1 : numDims;
    TORCH_CHECK(numDims <= MAX_DIMS, "input tensor has too many dimensions");
    int64_t sliceSize = self.dim() == 0 ? 1 : self.size(dim);

    auto input = self.contiguous();
    // static_cast is required to ensure that the correct type (INDEX_T)
    // is provided to the kernel for the arguments.
#define RUN_K(INDEX_T, DIM)                                      \
  pytorch_topk::launch<scalar_t, INDEX_T, DIM>(                  \
      inputInfo,                                                 \
      static_cast<INDEX_T>(sliceSize),                           \
      static_cast<INDEX_T>(k),                                   \
      largest,                                                   \
      static_cast<INDEX_T>(numInputSlices),                      \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]), \
      topKInfo,                                                  \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),   \
      indicesInfo,                                               \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));

#define RUN_DIM(INDEX_T) \
  if (allDims == 1)      \
  {                      \
    RUN_K(INDEX_T, 1);   \
  }                      \
  else if (allDims == 2) \
  {                      \
    RUN_K(INDEX_T, 2);   \
  }                      \
  else if (allDims == 3) \
  {                      \
    RUN_K(INDEX_T, 3);   \
  }                      \
  else                   \
  {                      \
    RUN_K(INDEX_T, -1);  \
  }

#define RUN_T(INDEX_T) \
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "topk_out_cuda", [&] { \
    at::cuda::detail::TensorInfo<const scalar_t, INDEX_T> inputInfo =     \
      at::cuda::detail::getTensorInfo<const scalar_t, INDEX_T>(input);    \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> topKInfo =            \
      at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(values);         \
    at::cuda::detail::TensorInfo<int64_t, INDEX_T> indicesInfo =          \
      at::cuda::detail::getTensorInfo<int64_t, INDEX_T>(indices);         \
    /* tensorInfoLegacyIfScalar*/                                         \
    if (!input.dim()) {                                                   \
      inputInfo.dims = 1;                                                 \
      inputInfo.sizes[0] = 1;                                             \
      inputInfo.strides[0] = 1;                                           \
      topKInfo.dims = 1;                                                  \
      topKInfo.sizes[0] = 1;                                              \
      topKInfo.strides[0] = 1;                                            \
      indicesInfo.dims = 1;                                               \
      indicesInfo.sizes[0] = 1;                                           \
      indicesInfo.strides[0] = 1;                                         \
    }                                                                     \
    /* We use these structures solely to find the offset to */            \
    /* each slice we are operating on */                                  \
    inputInfo.sizes[dim] = 1;                                             \
    topKInfo.sizes[dim] = 1;                                              \
    indicesInfo.sizes[dim] = 1;                                           \
    /* stash the stride of dim because it can be accidentally collapsed */ \
    auto strideTopK = topKInfo.strides[dim];                              \
    auto strideIndices = indicesInfo.strides[dim];                        \
    /* Collapse all other dims */                                         \
    int collapseInputDim = inputInfo.collapseDims(dim);                   \
    int collapseTopKDim = topKInfo.collapseDims(dim);                     \
    int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
    /* restore stride in case it was collapsed */                         \
    topKInfo.strides[collapseTopKDim] = strideTopK;                       \
    indicesInfo.strides[collapseIndicesDim] = strideIndices;              \
    int64_t numInputSlices = 1;                                           \
    for (int i = 0; i < inputInfo.dims; ++i) {                            \
      numInputSlices *= inputInfo.sizes[i];                               \
    }                                                                     \
                                                                          \
    /* This is used as a template parameter to calculate indices. */      \
    /* We only specialize it if all collapsed dim sizes are the */        \
    /* same; otherwise, we use -1 which is the specialization */          \
    /* parameter for arbitrary dimensions */                              \
    int allDims = inputInfo.dims;                                         \
    if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
      allDims = -1;                                                       \
    }                                                                     \
                                                                          \
    RUN_DIM(INDEX_T); });

    // the below is safe with 0-dimensional tensors because it is based on
    // TensorInfo which implicitly expands to 1-dimensional.
    if (input.numel() > 0)
    {
      // Based on required index size, run the algorithm with the
      // appropriate index type
      if (at::cuda::detail::canUse32BitIndexMath(input) &&
          at::cuda::detail::canUse32BitIndexMath(values) &&
          at::cuda::detail::canUse32BitIndexMath(indices))
      {
        RUN_T(uint32_t);
      }
      else
      {
        RUN_T(uint64_t);
      }
    }
#undef RUN_T
#undef RUN_DIM
  }

  PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
  {
    m.def("topk", &launch_gather_topk_kernel);
  }

} // pytorch_topk
