#include "hip/hip_runtime.h"

// #define TORCH_ASSERT_NO_OPERATORS
#include <ATen/core/TensorBase.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/ScanUtils.cuh>
#include <ATen/cuda/DeviceUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <limits>

#include <torch/extension.h>

#include <c10/macros/Macros.h>

namespace approx_topk
{
  using namespace at;
  using namespace at::native;

  constexpr int MAX_QUEUE_SIZE = 16;

  template <typename T>
  struct AddOp
  {
    __device__ __forceinline__ T operator()(T const &lhs, T const &rhs)
    {
      return (lhs + rhs);
    }
  };

  template <typename T, typename IndexType>
  __device__ void insertIntoQueues(
      T v, IndexType index,
      T *valueQueue, IndexType *indexQueue,
      IndexType j, bool largest)
  {
    // The smallest (or largest) item is at the start of the queue. We walk down the
    // queue inserting the new item (if possible), and shifting the existing, smaller,
    // items backwards.
    for (IndexType i = 0; i < j; i++)
    {
      if ((largest && valueQueue[i] > v) || (!largest && valueQueue[i] < v))
        break;
      if (i > 0)
      {
        valueQueue[i - 1] = valueQueue[i];
        indexQueue[i - 1] = indexQueue[i];
      }
      valueQueue[i] = v;
      indexQueue[i] = index;
    }
  }

  template <typename T, typename IndexType, int Dim>
  __global__ void priorityQueueTopK(
      at::cuda::detail::TensorInfo<const T, IndexType> input,
      IndexType inputSliceSize,
      IndexType k, // aka `k`
      IndexType j,
      bool largest,

      IndexType numInputSlices,
      IndexType inputWithinSliceStride,

      at::cuda::detail::TensorInfo<T, IndexType> topK,
      IndexType topKWithinSliceStride,

      at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
      IndexType indicesWithinSliceStride)
  {
    IndexType sliceIndex = blockIdx.x;
    IndexType bucketIndex = blockIdx.y + threadIdx.x;
    IndexType numBuckets = k == 0 ? 1 : k / j;
    if (sliceIndex >= numInputSlices || bucketIndex >= numBuckets)
    {
      return;
    }

    IndexType outputBucketSize = k / numBuckets;
    // If the number of buckets divides the input slice size then we just equally divide
    // the input slice between the buckets.
    // If the number of buckets does not exactly divide the slice size then we round the
    // bucket size down leaving some remainder, r, of the slice. In order to cover the
    // remainder, we increase the size of the first r buckets by one.
    IndexType baseInputBucketSize = inputSliceSize / numBuckets;
    IndexType remainder = inputSliceSize - baseInputBucketSize * numBuckets;
    IndexType inputBucketSize = baseInputBucketSize;
    if (bucketIndex < remainder)
    {
      inputBucketSize += 1;
    }

    IndexType previousBigBuckets = min(bucketIndex, remainder);
    IndexType previousNormalBuckets =
        bucketIndex > remainder ? bucketIndex - remainder : 0;
    IndexType inputBucketOffset =
        previousBigBuckets * (baseInputBucketSize + 1) + previousNormalBuckets * baseInputBucketSize;
    IndexType inputStartIndex =
        at::cuda::detail::IndexToOffset<const T, IndexType, Dim>::get(sliceIndex, input);
    inputStartIndex += inputBucketOffset;
    const T *inputStart = &input.data[inputStartIndex];

    T valueQueue[MAX_QUEUE_SIZE];
    IndexType indexQueue[MAX_QUEUE_SIZE];
    for (IndexType i = 0; i < j; i++)
    {
      if (largest)
        valueQueue[i] = std::numeric_limits<T>::lowest();
      else
        valueQueue[i] = std::numeric_limits<T>::max();
    }
    for (IndexType i = 0; i < inputBucketSize; i++)
    {
      T v = doLdg(&inputStart[i * inputWithinSliceStride]);
      insertIntoQueues(v, i, valueQueue, indexQueue, j, largest);
    }

    IndexType valuesOutputStartIndex =
        at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(sliceIndex, topK);
    IndexType indicesOutputStartIndex =
        at::cuda::detail::IndexToOffset<int64_t, IndexType, Dim>::get(sliceIndex, indices);
    valuesOutputStartIndex += bucketIndex * outputBucketSize;
    indicesOutputStartIndex += bucketIndex * outputBucketSize;
    T *valuesOutputStart = &topK.data[valuesOutputStartIndex];
    int64_t *indicesOutputStart = &indices.data[indicesOutputStartIndex];

    for (IndexType i = 0; i < j; i++)
    {
      IndexType topKOffset = i * topKWithinSliceStride;
      IndexType indexOffset = i * indicesWithinSliceStride;
      valuesOutputStart[topKOffset] = valueQueue[i];
      indicesOutputStart[indexOffset] = indexQueue[i] + inputBucketOffset;
    }
  };

  template <typename T, typename IndexType, int Dim>
  void launch(
      at::cuda::detail::TensorInfo<const T, IndexType> input,
      IndexType inputSliceSize,
      IndexType k, // aka `k`
      IndexType j,
      bool largest,

      IndexType numInputSlices,
      IndexType inputWithinSliceStride,

      at::cuda::detail::TensorInfo<T, IndexType> topK,
      IndexType topKWithinSliceStride,

      at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
      IndexType indicesWithinSliceStride)
  {
    // We use the x dimension of the grid for batches provided by the user.
    // There is then one thread per bucket. We group these into warps of 32, and put
    // each warp in its own block.
    // 2^31 - 1 = the max grid size in the x dimension, from compute capability 3.0.
    TORCH_INTERNAL_ASSERT(numInputSlices < 2 ^ 31 - 1, "Too many slices for topk");
    TORCH_CHECK(k <= MAX_QUEUE_SIZE, "topk k too big")
    TORCH_CHECK(j <= MAX_QUEUE_SIZE, "topk j too big")
    TORCH_CHECK(k == 0 || j > 0, "topk j must be > 0");
    TORCH_CHECK(k == 0 || k % j == 0, "topk j must divide k");
    TORCH_CHECK(k <= inputSliceSize, "topk k must not be larger than topk size");
    IndexType numBuckets = k == 0 ? 1 : k / j;
    int warp_size = at::cuda::warp_size();
    IndexType blockY = at::ceil_div((int64_t)numBuckets, (int64_t)warp_size);
    dim3 grid(numInputSlices, blockY, 1);
    dim3 block(warp_size);

    priorityQueueTopK<T, IndexType, Dim><<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(
        input,
        inputSliceSize,
        k,
        j,
        largest,
        numInputSlices,
        inputWithinSliceStride,
        topK,
        topKWithinSliceStride,
        indices,
        indicesWithinSliceStride);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  void launch_priority_queue_topk_kernel(
      const Tensor &self, int64_t k, int64_t j, int64_t dim, bool largest,
      const Tensor &values, const Tensor &indices)
  {
    TensorArg input_arg{self, "xs", 1}, topK_arg{values, "valuesOutput", 2},
        indices_arg{indices, "indicesOutput", 3};
    checkAllSameGPU(__func__, {input_arg, topK_arg, indices_arg});

    int numDims = self.dim();
    numDims = numDims == 0 ? 1 : numDims;
    TORCH_CHECK(numDims <= MAX_DIMS, "input tensor has too many dimensions");
    int64_t sliceSize = self.dim() == 0 ? 1 : self.size(dim);

    auto input = self.contiguous();
    // static_cast is required to ensure that the correct type (INDEX_T)
    // is provided to the kernel for the arguments.
#define RUN_K(INDEX_T, DIM)                                      \
  launch<scalar_t, INDEX_T, DIM>(                                \
      inputInfo,                                                 \
      static_cast<INDEX_T>(sliceSize),                           \
      static_cast<INDEX_T>(k),                                   \
      static_cast<INDEX_T>(j),                                   \
      largest,                                                   \
      static_cast<INDEX_T>(numInputSlices),                      \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]), \
      topKInfo,                                                  \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),   \
      indicesInfo,                                               \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));

#define RUN_DIM(INDEX_T) \
  if (allDims == 1)      \
  {                      \
    RUN_K(INDEX_T, 1);   \
  }                      \
  else if (allDims == 2) \
  {                      \
    RUN_K(INDEX_T, 2);   \
  }                      \
  else if (allDims == 3) \
  {                      \
    RUN_K(INDEX_T, 3);   \
  }                      \
  else                   \
  {                      \
    RUN_K(INDEX_T, -1);  \
  }

#define RUN_T(INDEX_T) \
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "topk_out_cuda", [&] { \
    at::cuda::detail::TensorInfo<const scalar_t, INDEX_T> inputInfo =     \
      at::cuda::detail::getTensorInfo<const scalar_t, INDEX_T>(input);    \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> topKInfo =            \
      at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(values);         \
    at::cuda::detail::TensorInfo<int64_t, INDEX_T> indicesInfo =          \
      at::cuda::detail::getTensorInfo<int64_t, INDEX_T>(indices);         \
    /* tensorInfoLegacyIfScalar*/                                         \
    if (!input.dim()) {                                                   \
      inputInfo.dims = 1;                                                 \
      inputInfo.sizes[0] = 1;                                             \
      inputInfo.strides[0] = 1;                                           \
      topKInfo.dims = 1;                                                  \
      topKInfo.sizes[0] = 1;                                              \
      topKInfo.strides[0] = 1;                                            \
      indicesInfo.dims = 1;                                               \
      indicesInfo.sizes[0] = 1;                                           \
      indicesInfo.strides[0] = 1;                                         \
    }                                                                     \
    /* We use these structures solely to find the offset to */            \
    /* each slice we are operating on */                                  \
    inputInfo.sizes[dim] = 1;                                             \
    topKInfo.sizes[dim] = 1;                                              \
    indicesInfo.sizes[dim] = 1;                                           \
    /* stash the stride of dim because it can be accidentally collapsed */ \
    auto strideTopK = topKInfo.strides[dim];                              \
    auto strideIndices = indicesInfo.strides[dim];                        \
    /* Collapse all other dims */                                         \
    int collapseInputDim = inputInfo.collapseDims(dim);                   \
    int collapseTopKDim = topKInfo.collapseDims(dim);                     \
    int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
    /* restore stride in case it was collapsed */                         \
    topKInfo.strides[collapseTopKDim] = strideTopK;                       \
    indicesInfo.strides[collapseIndicesDim] = strideIndices;              \
    int64_t numInputSlices = 1;                                           \
    for (int i = 0; i < inputInfo.dims; ++i) {                            \
      numInputSlices *= inputInfo.sizes[i];                               \
    }                                                                     \
                                                                          \
    /* This is used as a template parameter to calculate indices. */      \
    /* We only specialize it if all collapsed dim sizes are the */        \
    /* same; otherwise, we use -1 which is the specialization */          \
    /* parameter for arbitrary dimensions */                              \
    int allDims = inputInfo.dims;                                         \
    if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
      allDims = -1;                                                       \
    }                                                                     \
                                                                          \
    RUN_DIM(INDEX_T); });

    // the below is safe with 0-dimensional tensors because it is based on
    // TensorInfo which implicitly expands to 1-dimensional.
    if (input.numel() > 0)
    {
      // Based on required index size, run the algorithm with the
      // appropriate index type
      if (at::cuda::detail::canUse32BitIndexMath(input) &&
          at::cuda::detail::canUse32BitIndexMath(values) &&
          at::cuda::detail::canUse32BitIndexMath(indices))
      {
        RUN_T(uint32_t);
      }
      else
      {
        RUN_T(uint64_t);
      }
    }
#undef RUN_T
#undef RUN_DIM
  }

  PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
  {
    m.def("topk", &launch_priority_queue_topk_kernel);
  }

} // approx_topk
